#include <stdio.h>
// System includes
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <iostream>       // std::cout, std::endl
#include <thread>         // std::this_thread::sleep_for
#include <chrono>
#include <fstream>
// #include <torch/torch.h>

// CUDA runtime
#include <hip/hip_runtime.h>



// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  float x = (float)i;
  float s = sinf(x); 
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s*s+c*c);
}

float maxError(float *a, int n) 
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int blockSize = 256, nStreams = 400;
  const long int n = 4 * 1024 * blockSize * nStreams;
  const long int streamSize = n / nStreams;
  const long int streamBytes = streamSize * sizeof(float);
  const long int bytes = n * sizeof(float);
   
  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );
  
  // allocate pinned host memory and device memory
  float *a, *d_a;
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) );      // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device

  float ms; // elapsed time in milliseconds
  
  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  auto start_prof = std::chrono::high_resolution_clock::now();
  auto end_prof = std::chrono::high_resolution_clock::now();
  auto request_prof = std::chrono::duration_cast<std::chrono::microseconds>(end_prof-start_prof);

  
  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );

  start_prof = std::chrono::high_resolution_clock::now();
  checkCuda( hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) );
  end_prof = std::chrono::high_resolution_clock::now();
  request_prof = std::chrono::duration_cast<std::chrono::microseconds>(end_prof-start_prof);
  std::cout << request_prof.count() << "us : hipMemcpy duration1\n";

  kernel<<<n/blockSize, blockSize>>>(d_a, 0);

  start_prof = std::chrono::high_resolution_clock::now();
  checkCuda( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) );
  end_prof = std::chrono::high_resolution_clock::now();
  request_prof = std::chrono::duration_cast<std::chrono::microseconds>(end_prof-start_prof);
  std::cout << request_prof.count() << "us : hipMemcpy duration2\n";

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for sequential transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );

  
  for (int i = 0; i < nStreams; ++i) {
    
    int offset = i * streamSize;

    start_prof = std::chrono::high_resolution_clock::now();
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], 
                               streamBytes, hipMemcpyHostToDevice, 
                               stream[i]) );

    end_prof = std::chrono::high_resolution_clock::now();
    request_prof = std::chrono::duration_cast<std::chrono::microseconds>(end_prof-start_prof);
    std::cout << request_prof.count() << "us : hipMemcpyAsync duration version 1\n";


    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset], 
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }

  

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 2: 
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );

  start_prof = std::chrono::high_resolution_clock::now();

  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], 
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
  }

  end_prof = std::chrono::high_resolution_clock::now();
  request_prof = std::chrono::duration_cast<std::chrono::microseconds>(end_prof-start_prof);
  std::cout << request_prof.count() << "us : hipMemcpyAsync duration1 version 2\n";


  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
  }

  start_prof = std::chrono::high_resolution_clock::now();

  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset], 
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }

  end_prof = std::chrono::high_resolution_clock::now();
  request_prof = std::chrono::duration_cast<std::chrono::microseconds>(end_prof-start_prof);
  std::cout << request_prof.count() << "us : hipMemcpyAsync duration2 version 2\n";

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}